#include "hip/hip_runtime.h"
/* FIXME: Edit this file to complete the functionality of 2D separable 
 * convolution on the GPU. You may add additional kernel functions 
 * as necessary. 
 */

__global__ void convolve_rows_kernel_naive(float *result, float *input, float *kernel,\
                                        int num_cols, int num_rows, int half_width)
{
    int i, i1;
    int j, j1, j2;
    int x, y;
    
    x = blockIdx.x * blockDim.x + threadIdx.x;
    y = blockIdx.y * blockDim.y + threadIdx.y;
    
    j1 = x - half_width;
    j2 = x + half_width;
    /* Clamp at the edges of the matrix */
    if (j1 < 0) 
        j1 = 0;
    if (j2 >= num_cols) 
        j2 = num_cols - 1;

    /* Obtain relative position of starting element from element being convolved */
    i1 = j1 - x; 
    
    j1 = j1 - x + half_width; /* Obtain operating width of the kernel */
    j2 = j2 - x + half_width;

    /* Convolve along row */
    result[y * num_cols + x] = 0.0f;
    for(i = i1, j = j1; j <= j2; j++, i++)
        result[y * num_cols + x] += 
            kernel[j] * input[y * num_cols + x + i];
    
    return;
}

__global__ void convolve_columns_kernel_naive(float *result, float *input, float *kernel,\
                                            int num_cols, int num_rows, int half_width)
{
    int i, i1;
    int j, j1, j2;
    int x, y;

    x = blockIdx.x * blockDim.x + threadIdx.x;
    y = blockIdx.y * blockDim.y + threadIdx.y;

    j1 = y - half_width;
    j2 = y + half_width;
    /* Clamp at the edges of the matrix */
    if (j1 < 0) 
        j1 = 0;
    if (j2 >= num_rows) 
        j2 = num_rows - 1;

    /* Obtain relative position of starting element from element being convolved */
    i1 = j1 - y; 
    
    j1 = j1 - y + half_width; /* Obtain the operating width of the kernel.*/
    j2 = j2 - y + half_width;

    /* Convolve along column */            
    result[y * num_cols + x] = 0.0f;
    for (i = i1, j = j1; j <= j2; j++, i++)
        result[y * num_cols + x] += 
            kernel[j] * input[y * num_cols + x + (i * num_cols)];
    
    return;
}

__global__ void convolve_rows_kernel_optimized(float *result, float *input,\
                                            int num_cols, int num_rows, int half_width)
{
    __shared__ float input_s[THREAD_BLOCK_SIZE * (2 * HALF_WIDTH + THREAD_BLOCK_SIZE)];

    int i, i1;
    int j, j1, j2;
    int x, y;
    
    x = blockIdx.x * blockDim.x + threadIdx.x;
    y = blockIdx.y * blockDim.y + threadIdx.y;
    
    int shift = threadIdx.y * (2 * HALF_WIDTH + THREAD_BLOCK_SIZE);
    int left_halo_index = (blockIdx.x - 1) * blockDim.x + threadIdx.x;
    if (threadIdx.x >= (blockDim.x - half_width)) {
        if (left_halo_index < 0)
            input_s[threadIdx.x - (blockDim.x - half_width) + shift] = 0.0;
        else
            input_s[threadIdx.x - (blockDim.x - half_width) + shift] = input[left_halo_index + y * num_rows];
    }

    if (x < num_cols)			 
        input_s[half_width + threadIdx.x + shift] = input[x + y * num_rows];
    else 
        input_s[half_width + threadIdx.x + shift] = 0.0;

    int right_halo_index = (blockIdx.x + 1) * blockDim.x + threadIdx.x;
    if (threadIdx.x < half_width) {
        if (right_halo_index >= num_cols * num_rows) 
            input_s[threadIdx.x + (blockDim.x + half_width) + shift] = 0.0;
        else
            input_s[threadIdx.x + (blockDim.x + half_width) + shift] = input[right_halo_index + y * num_rows];
    }

    __syncthreads();

    j1 = x - half_width;
    j2 = x + half_width;
    /* Clamp at the edges of the matrix */
    if (j1 < 0) 
        j1 = 0;
    if (j2 >= num_cols) 
        j2 = num_cols - 1;

    /* Obtain relative position of starting element from element being convolved */
    i1 = j1 - x; 
    
    j1 = j1 - x + half_width; /* Obtain operating width of the kernel */
    j2 = j2 - x + half_width;

    /* Convolve along row */
    result[y * num_cols + x] = 0.0f;
    for(i = i1, j = j1; j <= j2; j++, i++)
        result[y * num_cols + x] += 
            kernel_c[j] * input_s[threadIdx.y * blockDim.y + threadIdx.x + i + half_width];
    
    return;
}

__global__ void convolve_columns_kernel_optimized(float *result, float *input,\
                                                int num_cols, int num_rows, int half_width)
{
    __shared__ float input_s[THREAD_BLOCK_SIZE * (2 * HALF_WIDTH + THREAD_BLOCK_SIZE)];

    int i, i1;
    int j, j1, j2;
    int x, y;
    
    x = blockIdx.x * blockDim.x + threadIdx.x;
    y = blockIdx.y * blockDim.y + threadIdx.y;
    
    int shift = THREAD_BLOCK_SIZE + threadIdx.x;
    int left_halo_index = (blockIdx.y - 1) * blockDim.y + threadIdx.y;
    if (threadIdx.y >= (blockDim.y - half_width)) {
        if (left_halo_index < 0)
            input_s[threadIdx.y - (blockDim.y - half_width) * shift] = 0.0;
        else
            input_s[threadIdx.y - (blockDim.y - half_width) * shift] = input[left_halo_index * num_rows + x];
    }

    if (y < num_rows)			 
        input_s[half_width + threadIdx.y * shift] = input[y * num_rows + x];
    else 
        input_s[half_width + threadIdx.y * shift] = 0.0;

    int right_halo_index = (blockIdx.y + 1) * blockDim.y + threadIdx.y;
    if (threadIdx.y < half_width) {
        if (right_halo_index >= num_cols * num_rows) 
            input_s[threadIdx.y + (blockDim.y + half_width) * shift] = 0.0;
        else
            input_s[threadIdx.y + (blockDim.y + half_width) * shift] = input[right_halo_index * num_rows + x];
    }

    __syncthreads();

    j1 = y - half_width;
    j2 = y + half_width;
    /* Clamp at the edges of the matrix */
    if (j1 < 0) 
        j1 = 0;
    if (j2 >= num_rows) 
        j2 = num_rows - 1;

    /* Obtain relative position of starting element from element being convolved */
    i1 = j1 - y; 
    
    j1 = j1 - y + half_width; /* Obtain the operating width of the kernel.*/
    j2 = j2 - y + half_width;

    /* Convolve along column */            
    result[y * num_cols + x] = 0.0f;
    for (i = i1, j = j1; j <= j2; j++, i++)
        result[y * num_cols + x] += 
            kernel_c[j] * input_s[(threadIdx.y + half_width) * blockDim.x + threadIdx.x + (i * blockDim.x)];

    return;
}